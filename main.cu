#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <limits>
#include <sstream>
#include <string>
#include <vector>

#define CHECK(call)                                                         \
    do {                                                                    \
        const hipError_t error_code = call;                                \
        if (error_code != hipSuccess) {                                    \
            printf("CUDA ERROR: \n");                                       \
            printf("    FILE: %s\n", __FILE__);                             \
            printf("    LINE: %d\n", __LINE__);                             \
            printf("    ERROR CODE: %d\n", error_code);                     \
            printf("    ERROR TEXT: %s\n", hipGetErrorString(error_code)); \
            exit(1);                                                        \
        }                                                                   \
    } while (0);

struct Point {
    double x;
    double y;
    Point() {}
    Point(double x_, double y_) : x(x_), y(y_) {}
};

struct BoundingBox {
    Point min;
    Point max;
    BoundingBox() {}
    BoundingBox(Point min_, Point max_) : max(max_), min(min_) {}
};

BoundingBox calculateBoundingBox(const std::vector<Point>& polygon) {
    BoundingBox bbox;
    bbox.min.x = std::numeric_limits<double>::max();
    bbox.min.y = std::numeric_limits<double>::max();
    bbox.max.x = std::numeric_limits<double>::min();
    bbox.max.y = std::numeric_limits<double>::min();

    for (const Point& vertex : polygon) {
        if (vertex.x < bbox.min.x) bbox.min.x = vertex.x;
        if (vertex.y < bbox.min.y) bbox.min.y = vertex.y;
        if (vertex.x > bbox.max.x) bbox.max.x = vertex.x;
        if (vertex.y > bbox.max.y) bbox.max.y = vertex.y;
    }

    return bbox;
}

std::vector<Point> extractCoordinates(const std::string& wkt) {
    std::vector<Point> polygonCoordinates;
    size_t start = wkt.find("((") + 2;
    size_t end = wkt.find("))");

    if (start != std::string::npos && end != std::string::npos) {
        std::string coordinatesStr = wkt.substr(start, end - start);  // 提取坐标部分
        std::istringstream iss(coordinatesStr);
        std::vector<double> coordinates;
        double coordinate;
        while (iss >> coordinate) {
            coordinates.push_back(coordinate);

            if (iss.peek() == ',') {
                iss.ignore();
                polygonCoordinates.push_back(Point(coordinates[0], coordinates[1]));
                coordinates.clear();
            }
        }
        if (!coordinates.empty()) {
            polygonCoordinates.push_back(Point(coordinates[0], coordinates[1]));
        }
    }

    return polygonCoordinates;
}

__global__ void pixelizeBoundingBoxKernel(const BoundingBox* bbox, int rows, int cols, Point* pixels) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < rows && j < cols) {
        double pixelWidth = (bbox->max.x - bbox->min.x) / cols;
        double pixelHeight = (bbox->max.y - bbox->min.y) / rows;
        // printf("pixelHeight %.3f %.3f ", bbox->max.x, bbox->min.x);
        pixels[i * cols + j].x = bbox->min.x + (j + 0.5) * pixelWidth;
        pixels[i * cols + j].y = bbox->min.y + (i + 0.5) * pixelHeight;
    }
}

__global__ void pointInPolygonKernel(const Point* pixels, const Point* vertices, int pointsNum, int rows, int cols, int* results) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("i=%d ", i);
    // printf("j=%d ", j);
    if (i < rows && j < cols) {
        double x = pixels[i * cols + j].x;
        double y = pixels[i * cols + j].y;

        int crossings = 0;
        // printf("enter");
        for (int k = 0; k < pointsNum; k++) {
            double x1 = vertices[k].x;
            double y1 = vertices[k].y;
            double x2 = vertices[(k + 1) % pointsNum].x;
            double y2 = vertices[(k + 1) % pointsNum].y;

            if (((y1 > y) != (y2 > y)) && (x < (x2 - x1) * (y - y1) / (y2 - y1) + x1)) {
                crossings++;
            }
        }

        if (crossings % 2 == 1) {
            results[i * cols + j] = 1;
        } else {
            results[i * cols + j] = 0;
            // printf("%d", results[i * cols + j]);
        }
    }
}

__global__ void output(int rows, int cols, int* results) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%d", results[i * cols + j]);
        }
    }
}

int main() {
    std::string filename = "../bigpolygon.wkt";
    std::ifstream inputFile(filename);
    if (!inputFile.is_open()) {
        std::cerr << "无法打开文件" << std::endl;
        return 1;
    }
    std::vector<Point> polygonCoordinates;
    std::string line;
    while (std::getline(inputFile, line)) {
        std::vector<Point> coordinates = extractCoordinates(line);
        polygonCoordinates.insert(polygonCoordinates.end(), coordinates.begin(), coordinates.end());
    }
    inputFile.close();

    BoundingBox bbox = calculateBoundingBox(polygonCoordinates);
    auto begin = std::chrono::high_resolution_clock::now();
    // 从host拷贝所有点数据到device
    Point* devicePolygon;
    int pointNum = polygonCoordinates.size();
    hipMalloc((void**)&devicePolygon, sizeof(Point) * pointNum);
    hipMemcpy(devicePolygon, polygonCoordinates.data(), sizeof(Point) * pointNum, hipMemcpyHostToDevice);
    // 从host拷贝bbox到device
    BoundingBox* deviceBbox;
    hipMalloc((void**)&deviceBbox, sizeof(BoundingBox));
    hipMemcpy(deviceBbox, &bbox, sizeof(BoundingBox), hipMemcpyHostToDevice);

    // 在gpu上执行pixelize
    Point* devicePixels;
    int rows = 20;
    int cols = 30;
    hipMalloc((void**)&devicePixels, rows * cols * sizeof(Point));
    // 在gpu上执行像素化
    dim3 pixelBlockSize(16, 16);
    dim3 pixelGridSize((cols + pixelBlockSize.x - 1) / pixelBlockSize.x, (rows + pixelBlockSize.y - 1) / pixelBlockSize.y);
    pixelizeBoundingBoxKernel<<<pixelGridSize, pixelBlockSize>>>(deviceBbox, rows, cols, devicePixels);

    Point* hostPixels = new Point[rows * cols];
    hipMemcpy(hostPixels, devicePixels, rows * cols * sizeof(Point), hipMemcpyDeviceToHost);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    int* hostResults = new int[rows * cols];
    // 在gpu上判断是否在多边形内
    int* deviceResults;
    hipMalloc((void**)&deviceResults, rows * cols * sizeof(int));
    pointInPolygonKernel<<<pixelGridSize, pixelBlockSize>>>(devicePixels, devicePolygon, pointNum, rows, cols, deviceResults);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    hipMemcpy(hostResults, deviceResults, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
    // output<<<1, 1>>>(rows, cols, deviceResults);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
    printf("Time measured: %.3f seconds.\n", elapsed.count() * 1e-9);

    // 输出像素点的坐标
    // for (int i = 0; i < rows; ++i) {
    //     for (int j = 0; j < cols; ++j) {
    //         if (hostResults[i * cols + j]) {
    //             Point pixel = hostPixels[i * cols + j];
    //             std::cout << "Pixel Center: (" << pixel.x << ", " << pixel.y << ")" << std::endl;
    //         }
    //     }
    // }

    hipDeviceSynchronize();
    // 释放设备内存
    hipFree(devicePolygon);
    hipFree(deviceBbox);
    hipFree(devicePixels);
    hipFree(deviceResults);
    free(hostPixels);
    free(hostResults);
    return 0;
}
